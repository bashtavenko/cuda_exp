#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hello_kernel.cuh"

// CUDA kernel implementation
__global__ void hello_world_kernel() {
  printf("Block %d, Thread %d: Hello world\n", blockIdx.x, threadIdx.x);
}

// Host function that launches the kernel
void launch_hello_kernel() {
  // Launch kernel with 2 blocks and 4 threads per block
  hello_world_kernel<<<2, 4>>>();

  // Wait for GPU to finish
  hipDeviceSynchronize();
}