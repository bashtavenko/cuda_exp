#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_utils.h"

constexpr int N = 10;

__global__ void add(int* a, int* b, int* c) {
  int tid = blockIdx.x;  // this thread handles the data at its thread id
  if (tid < N) c[tid] = a[tid] + b[tid];
}

int main(void) {
  int a[N];
  int b[N];
  int c[N];
  int* dev_a;
  int* dev_b;
  int* dev_c;

  // Allocate the memory on the GPU
  RETURN_IF_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
  RETURN_IF_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
  RETURN_IF_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

  // fill the arrays 'a' and 'b' on the CPU
  for (int i = 0; i < N; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }

  // Copy 'a' and 'b' to GPU
  RETURN_IF_ERROR(
      hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  RETURN_IF_ERROR(
      hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  // N blocks, 1 thread
  // N blocks x 1 thread = N parallel threads
  // We could have launched N/2 per block and 2 threads
  // or N/4 with 4, or 1 block and N threads
  add<<<N, 1>>>(dev_a, dev_b, dev_c);

  RETURN_IF_ERROR(
      hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  // display the results
  for (int i = 0; i < N; ++i) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  RETURN_IF_ERROR(hipFree(dev_a));
  RETURN_IF_ERROR(hipFree(dev_b));
  RETURN_IF_ERROR(hipFree(dev_c));

  return EXIT_SUCCESS;
}