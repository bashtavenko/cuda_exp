#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void) {
  // Just a dummy kernel
}

int main(void) {
  // Check for GPU
  int device_count;
  hipError_t err = hipGetDeviceCount(&device_count);
  if (err != hipSuccess) {
    printf("Failed to get CUDA device count: %s\n", hipGetErrorString(err));
    return EXIT_FAILURE;
  }
  printf("CUDA Device Count: %d\n", device_count);

  // Get device properties
  hipDeviceProp_t device_prop;
  err = hipGetDeviceProperties(&device_prop, 0);
  if (err != hipSuccess) {
    printf("Failed to get CUDA device properties: %s", hipGetErrorString(err));
    return EXIT_FAILURE;
  }

  // Launch kernel
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  printf("Launched kernel\n");
  return EXIT_SUCCESS;
}
