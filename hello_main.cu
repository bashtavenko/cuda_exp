#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_utils.h"

__global__ void dummy_kernel(void) {}

__device__ int add_device(int a, int b) { return a + b; }

__global__ void add(int a, int b, int* c) { *c = add_device(a, b); }

int main(void) {
  // Check for GPU
  int device_count;
  RETURN_IF_ERROR(hipGetDeviceCount(&device_count));

  printf("CUDA Device Count: %d\n", device_count);

  // Get device properties
  hipDeviceProp_t device_prop;
  RETURN_IF_ERROR(hipGetDeviceProperties(&device_prop, 0));

  // Launch basic kernel
  dummy_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  printf("Launched dummy kernel\n");

  // Launch addition
  int c;
  int* dev_c;
  RETURN_IF_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
  add<<<1, 1>>>(2, 7, dev_c);
  RETURN_IF_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
  printf("2 + 7 = %d\n", c);
  RETURN_IF_ERROR(hipFree(dev_c));

  return EXIT_SUCCESS;
}
