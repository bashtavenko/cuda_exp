#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_utils.h"

__global__ void dummy_kernel(void) {}

__device__ int add_device(int a, int b) { return a + b; }

__global__ void add(int a, int b, int* c) { *c = add_device(a, b); }

int main(void) {
  // Check for GPU
  int device_count;
  RETURN_IF_ERROR(hipGetDeviceCount(&device_count));
  printf("CUDA Device Count: %d\n", device_count);
  if (device_count == 0) {
    printf("No CUDA devices found!\n");
    return EXIT_FAILURE;
  }

  // Get device properties
  hipDeviceProp_t device_prop;
  RETURN_IF_ERROR(hipGetDeviceProperties(&device_prop, 0));

  // Launch basic kernel
  dummy_kernel<<<1, 1>>>();
  RETURN_IF_ERROR(hipGetLastError());
  RETURN_IF_ERROR(hipDeviceSynchronize());
  printf("Launched dummy kernel\n");

  // Launch addition
  int c;
  int* dev_c;
  RETURN_IF_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
  add<<<1, 1>>>(2, 7, dev_c);
  RETURN_IF_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
  printf("2 + 7 = %d\n", c);
  RETURN_IF_ERROR(hipFree(dev_c));

  return EXIT_SUCCESS;
}
